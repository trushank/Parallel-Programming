#include "hip/hip_runtime.h"
//******************************************************************************
//
// File:    PiGPU.cu
// Author:  Alan Kaminsky
// Version: 22-Oct-2013
//
// This source file is copyright (C) 2013 by Parallel Crypto LLC. All rights
// reserved. For further information, contact the author, Alan Kaminsky, at
// alan.kaminsky@parallelcrypto.com.
//
// This source file is free software; you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by the Free
// Software Foundation; either version 3 of the License, or (at your option) any
// later version.
//
// This source file is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
// FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
// details.
//
// A copy of the GNU General Public License is provided in the file gpl.txt. You
// may also obtain a copy of the GNU General Public License on the World Wide
// Web at http://www.gnu.org/licenses/gpl.html.
//
//******************************************************************************

/**
 * Program PiGPU computes an approximation of pi in parallel on the GPU by
 * generating N random (x,y) points in the unit square and counting how many
 * fall within a distance of 1 from the origin.
 *
 * Usage: PiGPU <seed> <N>
 * <seed> = Pseudorandom number generator seed
 * <N> = Number of points, N >= 1
 */

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

#include "Util.cu"
#include "Random.cu"

//------------------------------------------------------------------------------
// DEVICE FUNCTIONS

// Number of threads per block.
#define NT 1024

// Overall counter variable in global memory.
__device__ unsigned long long int devCount;

// Per-thread counter variables in shared memory.
__shared__ unsigned long long int shrCount [NT];

/**
 * Device kernel to compute random points.
 *
 * Called with a one-dimensional grid of one-dimensional blocks, NB blocks, NT
 * threads per block.
 *
 * @param  seed  Pseudorandom number generator seed.
 * @param  N     Number of points.
 */
__global__ void computeRandomPoints
	(unsigned long long int seed,
	 unsigned long long int N,unsigned long long int *devpopulation)
	{
	int x, size, rank;
	unsigned long long int len, lb, ub, count;
	prng_t prng;
	
	
	

	// Determine number of threads and this thread's rank.
	x = threadIdx.x;
	size = gridDim.x*NT;
	rank = blockIdx.x*NT + x;

	// Determine iterations for this thread.
	len = (N + size - 1)/size;
	lb = rank*len;
	ub = min (lb + len, N) - 1;

	// Initialize per-thread prng and count.
	prngSetSeed (&prng, seed + rank);
	count = 0;
	int sizeOfArray=(sizeof(&devpopulation) / sizeof(devpopulation[0]));
	unsigned long long int max=devpopulation[sizeOfArray-1];
	
	// Compute random points.
	for (unsigned long long int i = lb; i <= ub; ++ i)
		{
		int x = prngNextInt (&prng,max);
		int y = prngNextInt (&prng,max);
		
		if (x==y) y++;
		
		int xGrp=0;
		int yGrp=0;
		
			for(int i=0;i<sizeOfArray;i++){
				
				if(i==0){
					if(x>=0 && x<devpopulation[i]){
						xGrp=0;
					}
					if(y>=0 && y<devpopulation[i]){
					yGrp=0;
					}
				}
				else{
					if(x>=devpopulation[i-1] && x<devpopulation[i]){
					xGrp=i;
					}
					if(y>=devpopulation[i-1] && y<devpopulation[i]){
					yGrp=i;
					}
				}
			}
			if(xGrp!=yGrp) count++;
		}
// Shared memory parallel reduction within thread block.
	shrCount[x] = count;
	
	__syncthreads();
	for (int i = NT/2; i > 0; i >>= 1)
		{
		if (x < i)
			shrCount[x] += shrCount[x+i];
		__syncthreads();
		}

	// Atomic reduction into overall counter.
	if (x == 0)
		atomicAdd (&devCount, shrCount[0]);
	}

//------------------------------------------------------------------------------
// HOST FUNCTIONS

/**
 * Print a usage message and exit.
 */
static void usage()
	{
	fprintf (stderr, "Usage: PiGPU <seed> <N>\n");
	fprintf (stderr, "<seed> = Pseudorandom number generator seed\n");
	fprintf (stderr, "<trials> = Number of trials>= 1\n");
	exit (1);
	}

/**
 * Main program.
 */
int main
	(int argc,
	 char *argv[])
	{
	unsigned long long int seed, trials, t1, t2, hostCount;
	unsigned long long int *population,*devpopulation;
	int dev, NB;
   size_t populationBytes;
	// Parse command line arguments.
	if (argc < 4) usage();
	progname = argv[0];
	if (sscanf (argv[1], "%llu", &seed) != 1) usage();
	if (sscanf (argv[2], "%llu", &trials) != 1) usage();
	
	
populationBytes= (argc-3)*sizeof(unsigned long long int);
population=(unsigned long long int*) malloc (populationBytes);
 if (population == NULL) die ("Cannot allocate population");
 for(int i=3;i<argc;i++){
	if (sscanf (argv[i], "%llu", &population[i-3]) != 1 || population[i-3] < 1) usage();
	
	if(i!=3){
	population[i-3]+=population[i-4];
	}
	
}


	// Set CUDA device and determine number of multiprocessors (thread blocks).
	dev = setCudaDevice();
	checkCuda
		(hipDeviceGetAttribute (&NB, hipDeviceAttributeMultiprocessorCount, dev),
		 "Cannot get number of multiprocessors");
	printf ("NB = %d, NT = %d, threads = %d\n", NB, NT, NB*NT);

	// Allocate storage on host and device.
   checkCuda (hipMalloc (&devpopulation, populationBytes),
      "Cannot allocate devpopulation");
	  
	  // Copy population array to device.
   checkCuda (hipMemcpy (devpopulation, population, populationBytes, hipMemcpyHostToDevice),
      "Cannot upload devpopulation");
	  
	// Initialize overall counter.
	hostCount = 0;
	checkCuda
		(hipMemcpyToSymbol(HIP_SYMBOL(devCount), &hostCount, sizeof(hostCount)),
		 "Cannot initialize devCount");

	// Compute random points in parallel on the GPU. Measure computation time.
	t1 = currentTimeMillis();
	computeRandomPoints <<< NB, NT >>> (seed, trials,devpopulation);
	hipDeviceSynchronize();
	checkCuda
		(hipGetLastError(),
		 "Cannot launch computeRandomPoints() kernel");
	t2 = currentTimeMillis();

	// Get overall counter from GPU.
	checkCuda
		(hipMemcpyFromSymbol(&hostCount, HIP_SYMBOL(devCount), sizeof(hostCount)),
		 "Cannot copy devCount to hostCount");

	// Print results.
	printf ("pi = 100*%llu/%llu = %lf\n", hostCount, trials, 100*((double)hostCount/(double)trials));
	printf ("%llu msec\n", t2 - t1);
	}
